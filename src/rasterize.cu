#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya, Shuai Shao (Shrek)
 * @date      2012-2016
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <util/tiny_gltf_loader.h>
#include "rasterizeTools.h"
#include "rasterize.h"
#include <glm/gtc/quaternion.hpp>
#include <glm/gtc/matrix_transform.hpp>


#define TILE_SIZE 16
#define TILE_TRI_LIST_SCALE 0.5f
#define TRI_LIST_SCALE_THRESHOLD 100

#define ROUND_UP_DIV(x, n) (((x) + (n) - 1) / (n))


namespace {

	typedef unsigned short VertexIndex;
	typedef glm::vec3 VertexAttributePosition;
	typedef glm::vec3 VertexAttributeNormal;
	typedef glm::vec2 VertexAttributeTexcoord;
	typedef unsigned char TextureData;

	typedef unsigned char BufferByte;

	enum PrimitiveType{
		Point = 1,
		Line = 2,
		Triangle = 3
	};

	struct VertexOut {
		glm::vec4 pos;

		// TODO: add new attributes to your VertexOut
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		 glm::vec3 eyePos;	// eye space position used for shading
		 glm::vec3 eyeNor;	// eye space normal used for shading, cuz normal will go wrong after perspective transformation
		// glm::vec3 col;
		 glm::vec2 texcoord0;
		 TextureData* dev_diffuseTex = NULL;
		 int diffuseTexWidth;
		 int diffuseTexHeight;
		// ...
	};

	struct Primitive {
		PrimitiveType primitiveType = Triangle;	// C++ 11 init
		VertexOut v[3];
	};

	struct Fragment {
		//glm::vec3 color;

		// TODO: add new attributes to your Fragment
		// The attributes listed below might be useful, 
		// but always feel free to modify on your own

		glm::vec3 eyePos;	// eye space position used for shading
		glm::vec3 eyeNor;
		VertexAttributeTexcoord texcoord0;
		TextureData* dev_diffuseTex;
		int diffuseTexWidth;
		int diffuseTexHeight;
		int shouldShade;
		// ...
	};

	struct PrimitiveDevBufPointers {
		int primitiveMode;	//from tinygltfloader macro
		PrimitiveType primitiveType;
		int numPrimitives;
		int numIndices;
		int numVertices;

		// Vertex In, const after loaded
		VertexIndex* dev_indices;
		VertexAttributePosition* dev_position;
		VertexAttributeNormal* dev_normal;
		VertexAttributeTexcoord* dev_texcoord0;

		// Materials, add more attributes when needed
		int diffuseTexWidth;
		int diffuseTexHeight;
		TextureData* dev_diffuseTex;
		// TextureData* dev_specularTex;
		// TextureData* dev_normalTex;
		// ...

		// Vertex Out, vertex used for rasterization, this is changing every frame
		VertexOut* dev_verticesOut;

		// TODO: add more attributes when needed
	};

}

static std::map<std::string, std::vector<PrimitiveDevBufPointers>> mesh2PrimitivesMap;


static int width = 0;
static int height = 0;

static int totalNumPrimitives = 0;
static Primitive *dev_primitives = NULL;
static Fragment *dev_fragmentBuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;

static int * dev_depth = NULL;	// you might need this buffer when doing depth test

static int numTilesX = 0;
static int numTilesY = 0;
static int triListSize = 0;
static int *dev_primCounts = nullptr; // one entry per tile
static int *dev_tileTriLists = nullptr;


/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ 
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + y * w;
	int outIdx = w - x - 1 + (h - y - 1) * w;

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[outIdx].w = 0;
		pbo[outIdx].x = color.x;
		pbo[outIdx].y = color.y;
		pbo[outIdx].z = color.z;
    }
}

/** 
* Writes fragment colors to the framebuffer
*/
__global__
void render(int w, int h, Fragment *fragmentBuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h && fragmentBuffer[index].shouldShade) {
        //framebuffer[index] = fragmentBuffer[index].color;

		// TODO: add your fragment shader code here
		// Phone shading
		const glm::vec3 lightDir = glm::normalize(glm::vec3(1.f, 1.f, 1.f)); // in eye space
		const glm::vec3 ambientColor(1.f, 1.f, 1.f);
		glm::vec3 diffuseColor;
		const glm::vec3 specularColor(1.f, 1.f, 1.f);
		const float specExp = 20.f;
		const float Ka = 0.1f;
		const float Kd = 0.5f;
		const float Ks = 0.5f;

		if (fragmentBuffer[index].dev_diffuseTex)
		{
			int width = fragmentBuffer[index].diffuseTexWidth;
			int height = fragmentBuffer[index].diffuseTexHeight;
			glm::vec2 uv = fragmentBuffer[index].texcoord0;
			float fx = uv.x * width - 0.5f;
			float fy = uv.y * height - 0.5f;
			float wx = fx - glm::floor(fx);
			float wy = fy - glm::floor(fy);
			int x = glm::min(width - 2, glm::max(0, static_cast<int>(fx)));
			int y = glm::min(height - 2, glm::max(0, static_cast<int>(fy)));
			int i00 = y * width + x;
			int i10 = y * width + x + 1;
			int i01 = (y + 1) * width + x;
			int i11 = (y + 1) * width + x + 1;
			const TextureData *tex = fragmentBuffer[index].dev_diffuseTex;
			glm::vec3 p00(tex[i00 * 3] / 255.f, tex[i00 * 3 + 1] / 255.f, tex[i00 * 3 + 2] / 255.f);
			glm::vec3 p10(tex[i10 * 3] / 255.f, tex[i10 * 3 + 1] / 255.f, tex[i10 * 3 + 2] / 255.f);
			glm::vec3 p01(tex[i01 * 3] / 255.f, tex[i01 * 3 + 1] / 255.f, tex[i01 * 3 + 2] / 255.f);
			glm::vec3 p11(tex[i11 * 3] / 255.f, tex[i11 * 3 + 1] / 255.f, tex[i11 * 3 + 2] / 255.f);
			diffuseColor = (1.f - wy) * ((1.f - wx) * p00 + wx * p10) + wy * ((1.f - wx) * p01 + wx * p11);
		}
		else
		{
			diffuseColor = glm::vec3(1.f, 1.f, 1.f);
		}

		glm::vec3 eyePos = fragmentBuffer[index].eyePos;
		glm::vec3 eyeNor = fragmentBuffer[index].eyeNor;

		glm::vec3 h = glm::normalize(lightDir + glm::vec3(0.f, 0.f, 1.f));
		float costhetah = glm::max(0.f, glm::dot(h, eyeNor));
		float costheta = glm::max(0.f, glm::dot(lightDir, eyeNor));

		framebuffer[index] =
			Ka * ambientColor +
			Kd * diffuseColor * costheta +
			Ks * specularColor * powf(costhetah, specExp);
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	hipFree(dev_fragmentBuffer);
	hipMalloc(&dev_fragmentBuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height * sizeof(int));

	numTilesX = ROUND_UP_DIV(w, TILE_SIZE);
	numTilesY = ROUND_UP_DIV(h, TILE_SIZE);
	int numTiles = numTilesX * numTilesY;
	
	hipFree(dev_primCounts);
	hipMalloc(&dev_primCounts, numTiles * sizeof(int));
	hipFree(dev_tileTriLists);

	checkCUDAError("rasterizeInit");
}

__global__
void initDepth(int w, int h, int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
	{
		int index = x + (y * w);
		depth[index] = INT_MAX;
	}
}


/**
* kern function with support for stride to sometimes replace hipMemcpy
* One thread is responsible for copying one component
*/
__global__ 
void _deviceBufferCopy(int N, BufferByte* dev_dst, const BufferByte* dev_src, int n, int byteStride, int byteOffset, int componentTypeByteSize) {
	
	// Attribute (vec3 position)
	// component (3 * float)
	// byte (4 * byte)

	// id of component
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < N) {
		int count = i / n;
		int offset = i - count * n;	// which component of the attribute

		for (int j = 0; j < componentTypeByteSize; j++) {
			
			dev_dst[count * componentTypeByteSize * n 
				+ offset * componentTypeByteSize 
				+ j]

				= 

			dev_src[byteOffset 
				+ count * (byteStride == 0 ? componentTypeByteSize * n : byteStride) 
				+ offset * componentTypeByteSize 
				+ j];
		}
	}
	

}

__global__
void _nodeMatrixTransform(
	int numVertices,
	VertexAttributePosition* position,
	VertexAttributeNormal* normal,
	glm::mat4 MV, glm::mat3 MV_normal) {

	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {
		position[vid] = glm::vec3(MV * glm::vec4(position[vid], 1.0f));
		normal[vid] = glm::normalize(MV_normal * normal[vid]);
	}
}

glm::mat4 getMatrixFromNodeMatrixVector(const tinygltf::Node & n) {
	
	glm::mat4 curMatrix(1.0);

	const std::vector<double> &m = n.matrix;
	if (m.size() > 0) {
		// matrix, copy it

		for (int i = 0; i < 4; i++) {
			for (int j = 0; j < 4; j++) {
				curMatrix[i][j] = (float)m.at(4 * i + j);
			}
		}
	} else {
		// no matrix, use rotation, scale, translation

		if (n.translation.size() > 0) {
			curMatrix[3][0] = n.translation[0];
			curMatrix[3][1] = n.translation[1];
			curMatrix[3][2] = n.translation[2];
		}

		if (n.rotation.size() > 0) {
			glm::mat4 R;
			glm::quat q;
			q[0] = n.rotation[0];
			q[1] = n.rotation[1];
			q[2] = n.rotation[2];

			R = glm::mat4_cast(q);
			curMatrix = curMatrix * R;
		}

		if (n.scale.size() > 0) {
			curMatrix = curMatrix * glm::scale(glm::vec3(n.scale[0], n.scale[1], n.scale[2]));
		}
	}

	return curMatrix;
}

void traverseNode (
	std::map<std::string, glm::mat4> & n2m,
	const tinygltf::Scene & scene,
	const std::string & nodeString,
	const glm::mat4 & parentMatrix
	) 
{
	const tinygltf::Node & n = scene.nodes.at(nodeString);
	glm::mat4 M = parentMatrix * getMatrixFromNodeMatrixVector(n);
	n2m.insert(std::pair<std::string, glm::mat4>(nodeString, M));

	auto it = n.children.begin();
	auto itEnd = n.children.end();

	for (; it != itEnd; ++it) {
		traverseNode(n2m, scene, *it, M);
	}
}

void rasterizeSetBuffers(const tinygltf::Scene & scene) {

	totalNumPrimitives = 0;

	std::map<std::string, BufferByte*> bufferViewDevPointers;

	// 1. copy all `bufferViews` to device memory
	{
		std::map<std::string, tinygltf::BufferView>::const_iterator it(
			scene.bufferViews.begin());
		std::map<std::string, tinygltf::BufferView>::const_iterator itEnd(
			scene.bufferViews.end());

		for (; it != itEnd; it++) {
			const std::string key = it->first;
			const tinygltf::BufferView &bufferView = it->second;
			if (bufferView.target == 0) {
				continue; // Unsupported bufferView.
			}

			const tinygltf::Buffer &buffer = scene.buffers.at(bufferView.buffer);

			BufferByte* dev_bufferView;
			hipMalloc(&dev_bufferView, bufferView.byteLength);
			hipMemcpy(dev_bufferView, &buffer.data.front() + bufferView.byteOffset, bufferView.byteLength, hipMemcpyHostToDevice);

			checkCUDAError("Set BufferView Device Mem");

			bufferViewDevPointers.insert(std::make_pair(key, dev_bufferView));

		}
	}



	// 2. for each mesh: 
	//		for each primitive: 
	//			build device buffer of indices, materail, and each attributes
	//			and store these pointers in a map
	{

		std::map<std::string, glm::mat4> nodeString2Matrix;
		auto rootNodeNamesList = scene.scenes.at(scene.defaultScene);

		{
			auto it = rootNodeNamesList.begin();
			auto itEnd = rootNodeNamesList.end();
			for (; it != itEnd; ++it) {
				traverseNode(nodeString2Matrix, scene, *it, glm::mat4(1.0f));
			}
		}


		// parse through node to access mesh

		auto itNode = nodeString2Matrix.begin();
		auto itEndNode = nodeString2Matrix.end();
		for (; itNode != itEndNode; ++itNode) {

			const tinygltf::Node & N = scene.nodes.at(itNode->first);
			const glm::mat4 & matrix = itNode->second;
			const glm::mat3 & matrixNormal = glm::transpose(glm::inverse(glm::mat3(matrix)));

			auto itMeshName = N.meshes.begin();
			auto itEndMeshName = N.meshes.end();

			for (; itMeshName != itEndMeshName; ++itMeshName)
			{
				const tinygltf::Mesh & mesh = scene.meshes.at(*itMeshName);

				auto res = mesh2PrimitivesMap.insert(std::pair<std::string, std::vector<PrimitiveDevBufPointers>>(mesh.name, std::vector<PrimitiveDevBufPointers>()));
				std::vector<PrimitiveDevBufPointers> & primitiveVector = (res.first)->second;

				// for each primitive
				for (size_t i = 0; i < mesh.primitives.size(); i++)
				{
					const tinygltf::Primitive &primitive = mesh.primitives[i];

					if (primitive.indices.empty())
						return;

					// TODO: add new attributes for your PrimitiveDevBufPointers when you add new attributes
					VertexIndex* dev_indices = nullptr;
					VertexAttributePosition* dev_position = nullptr;
					VertexAttributeNormal* dev_normal = nullptr;
					VertexAttributeTexcoord* dev_texcoord0 = nullptr;

					// ----------Indices-------------

					const tinygltf::Accessor &indexAccessor = scene.accessors.at(primitive.indices);
					const tinygltf::BufferView &bufferView = scene.bufferViews.at(indexAccessor.bufferView);
					BufferByte* dev_bufferView = bufferViewDevPointers.at(indexAccessor.bufferView);

					// assume type is SCALAR for indices
					int n = 1;
					int numIndices = indexAccessor.count;
					int componentTypeByteSize = sizeof(VertexIndex);
					int byteLength = numIndices * n * componentTypeByteSize;

					dim3 numThreadsPerBlock(128);
					dim3 numBlocks((numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					hipMalloc(&dev_indices, byteLength);
					_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
						numIndices,
						(BufferByte*)dev_indices,
						dev_bufferView,
						n,
						indexAccessor.byteStride,
						indexAccessor.byteOffset,
						componentTypeByteSize);


					checkCUDAError("Set Index Buffer");


					// ---------Primitive Info-------

					// Warning: LINE_STRIP is not supported in tinygltfloader
					int numPrimitives;
					PrimitiveType primitiveType;
					switch (primitive.mode) {
					case TINYGLTF_MODE_TRIANGLES:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices / 3;
						break;
					case TINYGLTF_MODE_TRIANGLE_STRIP:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_TRIANGLE_FAN:
						primitiveType = PrimitiveType::Triangle;
						numPrimitives = numIndices - 2;
						break;
					case TINYGLTF_MODE_LINE:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices / 2;
						break;
					case TINYGLTF_MODE_LINE_LOOP:
						primitiveType = PrimitiveType::Line;
						numPrimitives = numIndices + 1;
						break;
					case TINYGLTF_MODE_POINTS:
						primitiveType = PrimitiveType::Point;
						numPrimitives = numIndices;
						break;
					default:
						// output error
						break;
					};


					// ----------Attributes-------------

					auto it(primitive.attributes.begin());
					auto itEnd(primitive.attributes.end());

					int numVertices = 0;
					// for each attribute
					for (; it != itEnd; it++) {
						const tinygltf::Accessor &accessor = scene.accessors.at(it->second);
						const tinygltf::BufferView &bufferView = scene.bufferViews.at(accessor.bufferView);

						int n = 1;
						if (accessor.type == TINYGLTF_TYPE_SCALAR) {
							n = 1;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC2) {
							n = 2;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC3) {
							n = 3;
						}
						else if (accessor.type == TINYGLTF_TYPE_VEC4) {
							n = 4;
						}

						BufferByte * dev_bufferView = bufferViewDevPointers.at(accessor.bufferView);
						BufferByte ** dev_attribute = NULL;

						numVertices = accessor.count;
						int componentTypeByteSize;

						// Note: since the type of our attribute array (dev_position) is static (float32)
						// We assume the glTF model attribute type are 5126(FLOAT) here

						if (it->first.compare("POSITION") == 0) {
							componentTypeByteSize = sizeof(VertexAttributePosition) / n;
							dev_attribute = (BufferByte**)&dev_position;
						}
						else if (it->first.compare("NORMAL") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeNormal) / n;
							dev_attribute = (BufferByte**)&dev_normal;
						}
						else if (it->first.compare("TEXCOORD_0") == 0) {
							componentTypeByteSize = sizeof(VertexAttributeTexcoord) / n;
							dev_attribute = (BufferByte**)&dev_texcoord0;
						}

						std::cout << accessor.bufferView << "  -  " << it->second << "  -  " << it->first << '\n';

						dim3 numThreadsPerBlock(128);
						dim3 numBlocks((n * numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
						int byteLength = numVertices * n * componentTypeByteSize;
						hipMalloc(dev_attribute, byteLength);

						_deviceBufferCopy << <numBlocks, numThreadsPerBlock >> > (
							n * numVertices,
							*dev_attribute,
							dev_bufferView,
							n,
							accessor.byteStride,
							accessor.byteOffset,
							componentTypeByteSize);

						std::string msg = "Set Attribute Buffer: " + it->first;
						checkCUDAError(msg.c_str());
					}

					// malloc for VertexOut
					VertexOut* dev_vertexOut;
					hipMalloc(&dev_vertexOut, numVertices * sizeof(VertexOut));
					checkCUDAError("Malloc VertexOut Buffer");

					// ----------Materials-------------

					// You can only worry about this part once you started to 
					// implement textures for your rasterizer
					TextureData* dev_diffuseTex = NULL;
					int diffuseTexWidth = 0, diffuseTexHeight = 0;

					if (!primitive.material.empty())
					{
						const tinygltf::Material &mat = scene.materials.at(primitive.material);
						printf("material.name = %s\n", mat.name.c_str());

						if (mat.values.find("diffuse") != mat.values.end()) {
							std::string diffuseTexName = mat.values.at("diffuse").string_value;
							if (scene.textures.find(diffuseTexName) != scene.textures.end()) {
								const tinygltf::Texture &tex = scene.textures.at(diffuseTexName);
								if (scene.images.find(tex.source) != scene.images.end()) {
									const tinygltf::Image &image = scene.images.at(tex.source);

									size_t s = image.image.size() * sizeof(TextureData);
									hipMalloc(&dev_diffuseTex, s);
									hipMemcpy(dev_diffuseTex, &image.image.at(0), s, hipMemcpyHostToDevice);
									
									// TODO: store the image size to your PrimitiveDevBufPointers
									diffuseTexWidth = image.width;
									diffuseTexHeight = image.height;

									checkCUDAError("Set Texture Image data");
								}
							}
						}

						// TODO: write your code for other materails
						// You may have to take a look at tinygltfloader
						// You can also use the above code loading diffuse material as a start point 
					}


					// ---------Node hierarchy transform--------
					hipDeviceSynchronize();
					
					dim3 numBlocksNodeTransform((numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
					_nodeMatrixTransform << <numBlocksNodeTransform, numThreadsPerBlock >> > (
						numVertices,
						dev_position,
						dev_normal,
						matrix,
						matrixNormal);

					checkCUDAError("Node hierarchy transformation");

					// at the end of the for loop of primitive
					// push dev pointers to map
					primitiveVector.push_back(PrimitiveDevBufPointers{
						primitive.mode,
						primitiveType,
						numPrimitives,
						numIndices,
						numVertices,

						dev_indices,
						dev_position,
						dev_normal,
						dev_texcoord0,

						diffuseTexWidth,
						diffuseTexHeight,
						dev_diffuseTex,

						dev_vertexOut	//VertexOut
					});

					totalNumPrimitives += numPrimitives;

				} // for each primitive

			} // for each mesh

		} // for each node

	}
	

	// 3. Malloc for dev_primitives
	{
		hipMalloc(&dev_primitives, totalNumPrimitives * sizeof(Primitive));
		int numTiles = numTilesX * numTilesY;
		if (totalNumPrimitives > TRI_LIST_SCALE_THRESHOLD)
		{
			triListSize = static_cast<int>(TILE_TRI_LIST_SCALE * totalNumPrimitives);
		}
		else
		{
			triListSize = totalNumPrimitives;
		}
		hipMalloc(&dev_tileTriLists, numTiles * triListSize * sizeof(int));
	}
	

	// Finally, hipFree raw dev_bufferViews
	{

		std::map<std::string, BufferByte*>::const_iterator it(bufferViewDevPointers.begin());
		std::map<std::string, BufferByte*>::const_iterator itEnd(bufferViewDevPointers.end());
			
			//bufferViewDevPointers

		for (; it != itEnd; it++) {
			hipFree(it->second);
		}

		checkCUDAError("Free BufferView Device Mem");
	}


}



__global__ 
void _vertexTransformAndAssembly(
	int numVertices, 
	PrimitiveDevBufPointers primitive, 
	glm::mat4 MVP, glm::mat4 MV, glm::mat3 MV_normal, 
	int width, int height)
{
	// vertex id
	int vid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (vid < numVertices) {

		// TODO: Apply vertex transformation here
		// Multiply the MVP matrix for each vertex position, this will transform everything into clipping space
		// Then divide the pos by its w element to transform into NDC space
		// Finally transform x and y to viewport space
		glm::vec3 pos = primitive.dev_position[vid];
		glm::vec3 nrm = primitive.dev_normal[vid];

		glm::vec3 eyePos = glm::vec3(MV * glm::vec4(pos, 1.f));
		glm::vec3 eyeNor = glm::normalize(MV_normal * nrm);

		glm::vec4 clipPos = MVP * glm::vec4(pos, 1.f);
		glm::vec4 outPos(
			(clipPos.x / clipPos.w + 1.f) * .5f * static_cast<float>(width),
			(clipPos.y / clipPos.w + 1.f) * .5f * static_cast<float>(height),
			clipPos.z,
			clipPos.w);

		// TODO: Apply vertex assembly here
		// Assemble all attribute arraies into the primitive array
		primitive.dev_verticesOut[vid].pos = outPos; // x, y in screen space. z in NDC
		primitive.dev_verticesOut[vid].eyePos = eyePos;
		primitive.dev_verticesOut[vid].eyeNor = eyeNor;
		if (primitive.dev_texcoord0)
		{
			primitive.dev_verticesOut[vid].texcoord0 = primitive.dev_texcoord0[vid];
		}
		if (primitive.dev_diffuseTex)
		{
			primitive.dev_verticesOut[vid].dev_diffuseTex = primitive.dev_diffuseTex;
		}
		primitive.dev_verticesOut[vid].diffuseTexWidth = primitive.diffuseTexWidth;
		primitive.dev_verticesOut[vid].diffuseTexHeight = primitive.diffuseTexHeight;
	}
}



static int curPrimitiveBeginId = 0;

__global__ 
void _primitiveAssembly(int numIndices, int curPrimitiveBeginId, Primitive* dev_primitives, PrimitiveDevBufPointers primitive) {

	// index id
	int iid = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (iid < numIndices) {

		// TODO: uncomment the following code for a start
		// This is primitive assembly for triangles

		int pid;	// id for cur primitives vector
		if (primitive.primitiveMode == TINYGLTF_MODE_TRIANGLES)
		{
			pid = iid / (int)primitive.primitiveType;
			dev_primitives[pid + curPrimitiveBeginId].v[iid % (int)primitive.primitiveType]
				= primitive.dev_verticesOut[primitive.dev_indices[iid]];
		}


		// TODO: other primitive types (point, line)
	}
	
}


__global__ void _rasterize(int numPrims, int width, int height, const Primitive *primitives, int *depthBuff, Fragment *fragments)
{
	int pid = blockDim.x * blockIdx.x + threadIdx.x;

	if (pid < numPrims)
	{
		const Primitive &prim = primitives[pid];

		glm::vec3 tri[3] =
		{
			glm::vec3(prim.v[0].pos),
			glm::vec3(prim.v[1].pos),
			glm::vec3(prim.v[2].pos)
		};
		AABB bbox = getAABBForTriangle(tri);

		int xmin = glm::min(width - 1, glm::max(0, static_cast<int>(bbox.min.x)));
		int xmax = glm::min(width - 1, glm::max(0, static_cast<int>(bbox.max.x)));
		int ymin = glm::min(height - 1, glm::max(0, static_cast<int>(bbox.min.y)));
		int ymax = glm::min(height - 1, glm::max(0, static_cast<int>(bbox.max.y)));

		for (int x = xmin; x <= xmax; ++x)
		{
			for (int y = ymin; y <= ymax; ++y)
			{
				glm::vec2 pix(x, y);
				glm::vec3 abc = calculateBarycentricCoordinate(tri, pix);

				if (isBarycentricCoordInBounds(abc))
				{
					// TODO
					// write fragment (x, y) if it passes depth test
					// For persepctive correct interpolation, we need to interpolate the reciprocal of
					// vertex depths before doing perspective division in order to obtain the correct
					// depth values. Interpolation of other vertex attributes also need special treatment
					// rather than iterpolating using screen-space Barycentric coordinates directly.
					// clipPos.w should be used instead of clipPos.z because, clipPos.w is the vertex's
					// depth in eye space multiplied by a constant. When it is used for interpolation,
					// the constant will eventually being cancelled out. Nonetheless, clipPos.z equals
					// the vertex's depth in eye space times a constant and then offset by -1. As a
					// result, it is not porpotional to the actual depth value of the vertex.
					int idx = y * width + x;
					float oneOverZ0 = 1.f / prim.v[0].pos.w;
					float oneOverZ1 = 1.f / prim.v[1].pos.w;
					float oneOverZ2 = 1.f / prim.v[2].pos.w;
					float oneOverPixDepth = getFloatAtCoordinate(abc, oneOverZ0, oneOverZ1, oneOverZ2);
					float pixDepth = 1.f / oneOverPixDepth;
					float pixDepthNDC = getFloatAtCoordinate(abc, tri[0].z * oneOverZ0, tri[1].z * oneOverZ1, tri[2].z * oneOverZ2);

					if (pixDepthNDC > -1.f && pixDepthNDC < 1.f)
					{
						int iPixDepth = static_cast<int>(pixDepthNDC * INT_MAX);
						int iOldDepth = atomicMin(&depthBuff[idx], iPixDepth);

						if (iPixDepth < iOldDepth)
						{
							//fragments[idx].color = glm::vec3(1.f, 1.f, 1.f);
							fragments[idx].eyePos =
								getVec3AtCoordinate(abc, prim.v[0].eyePos * oneOverZ0, prim.v[1].eyePos * oneOverZ1, prim.v[2].eyePos * oneOverZ2) * pixDepth;
							fragments[idx].eyeNor = glm::normalize(
								getVec3AtCoordinate(abc, prim.v[0].eyeNor * oneOverZ0, prim.v[1].eyeNor * oneOverZ1, prim.v[2].eyeNor * oneOverZ2) * pixDepth);
							if (prim.v[0].dev_diffuseTex)
							{
								fragments[idx].texcoord0 =
									getVec2AtCoordinate(abc, prim.v[0].texcoord0 * oneOverZ0, prim.v[1].texcoord0 * oneOverZ1, prim.v[2].texcoord0 * oneOverZ2) * pixDepth;
								fragments[idx].dev_diffuseTex =
									prim.v[0].dev_diffuseTex;
								fragments[idx].diffuseTexWidth = prim.v[0].diffuseTexWidth;
								fragments[idx].diffuseTexHeight = prim.v[0].diffuseTexHeight;
							}
							fragments[idx].shouldShade = 1;
						}
					}
				}
			}
		}
	}
}


__global__ void fillTileTriLists(int numPrims, int numTilesX, int numTilesY, int triListSize, const Primitive *primitives, int *primCounts, int *tileTriLists)
{
	int pidx = blockDim.x * blockIdx.x + threadIdx.x;

	if (pidx < numPrims)
	{
		const Primitive &prim = primitives[pidx];
		glm::vec3 tri[3] =
		{
			glm::vec3(prim.v[0].pos),
			glm::vec3(prim.v[1].pos),
			glm::vec3(prim.v[2].pos)
		};

		if (!isFrontFacing(tri))
		{
			return;
		}

		AABB bbox = getAABBForTriangle(tri);

		int tileXMin = glm::max(0, static_cast<int>(floorf((bbox.min.x + .5f) / TILE_SIZE)));
		int tileXMax = glm::min(numTilesX - 1, static_cast<int>(floorf((bbox.max.x + .5f) / TILE_SIZE)));
		int tileYMin = glm::max(0, static_cast<int>(floorf((bbox.min.y + .5f) / TILE_SIZE)));
		int tileYMax = glm::min(numTilesY - 1, static_cast<int>(floorf((bbox.max.y + .5f) / TILE_SIZE)));

		for (int y = tileYMin; y <= tileYMax; ++y)
		{
			for (int x = tileXMin; x <= tileXMax; ++x)
			{
				AABB tileBound =
				{
					{ float(x * TILE_SIZE) - .5f, float(y * TILE_SIZE) - .5f, 0.f },
					{ float((x + 1) * TILE_SIZE) - .5f, float((y + 1) * TILE_SIZE) - .5f, 0.f },
				};

				if (triAABBIntersect(tileBound, tri))
				{
					int tidx = y * numTilesX + x;
					int offset = atomicAdd(&primCounts[tidx], 1);
					tileTriLists[tidx * triListSize + offset] = pidx;
				}
			}
		}
	}
}


__global__ void tileBasedRasterize(
	int numPrims, int width, int height, int numTilesX, int numTilesY, int triListSize,
	const Primitive *primitives, int *primCounts, int *tileTriLists,
	int *depthBuff, Fragment *fragments)
{
	float l_depthBuff = 1.f;
	Fragment l_fragment{};

	int pixIdxX = blockIdx.x * TILE_SIZE + threadIdx.x;
	int pixIdxY = blockIdx.y * TILE_SIZE + threadIdx.y;
	int pixIdx = pixIdxY * width + pixIdxX;
	int tileIdx = blockIdx.y * numTilesX + blockIdx.x;
	int numTris = primCounts[tileIdx];
	int *triList = tileTriLists + triListSize * tileIdx;

	if (pixIdxX >= width || pixIdxY >= height)
	{
		return;
	}

	for (int i = 0; i < numTris; ++i)
	{
		const Primitive &prim = primitives[triList[i]];
		glm::vec3 tri[3] =
		{
			glm::vec3(prim.v[0].pos),
			glm::vec3(prim.v[1].pos),
			glm::vec3(prim.v[2].pos)
		};
		glm::vec3 abc = calculateBarycentricCoordinate(tri, glm::vec2(pixIdxX, pixIdxY));

		if (isBarycentricCoordInBounds(abc))
		{
			float oneOverZ0 = 1.f / prim.v[0].pos.w;
			float oneOverZ1 = 1.f / prim.v[1].pos.w;
			float oneOverZ2 = 1.f / prim.v[2].pos.w;
			float oneOverPixDepth = getFloatAtCoordinate(abc, oneOverZ0, oneOverZ1, oneOverZ2);
			float pixDepth = 1.f / oneOverPixDepth;
			float pixDepthNDC = getFloatAtCoordinate(abc, tri[0].z * oneOverZ0, tri[1].z * oneOverZ1, tri[2].z * oneOverZ2);

			if (pixDepthNDC > -1.f && pixDepthNDC < l_depthBuff)
			{
				l_depthBuff = pixDepthNDC;
				l_fragment.eyePos =
					getVec3AtCoordinate(abc, prim.v[0].eyePos * oneOverZ0, prim.v[1].eyePos * oneOverZ1, prim.v[2].eyePos * oneOverZ2) * pixDepth;
				l_fragment.eyeNor = glm::normalize(
					getVec3AtCoordinate(abc, prim.v[0].eyeNor * oneOverZ0, prim.v[1].eyeNor * oneOverZ1, prim.v[2].eyeNor * oneOverZ2) * pixDepth);
				l_fragment.texcoord0 =
					getVec2AtCoordinate(abc, prim.v[0].texcoord0 * oneOverZ0, prim.v[1].texcoord0 * oneOverZ1, prim.v[2].texcoord0 * oneOverZ2) * pixDepth;
				l_fragment.dev_diffuseTex =
					prim.v[0].dev_diffuseTex;
				l_fragment.diffuseTexWidth = prim.v[0].diffuseTexWidth;
				l_fragment.diffuseTexHeight = prim.v[0].diffuseTexHeight;
				l_fragment.shouldShade = true;
			}
		}
	}

	depthBuff[pixIdx] = static_cast<int>(l_depthBuff * INT_MAX);
	fragments[pixIdx] = l_fragment;
}


/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, const glm::mat4 & MVP, const glm::mat4 & MV, const glm::mat3 MV_normal) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);

	// Execute your rasterization pipeline here
	// (See README for rasterization pipeline outline.)

	// Vertex Process & primitive assembly
	{
		curPrimitiveBeginId = 0;
		dim3 numThreadsPerBlock(128);

		auto it = mesh2PrimitivesMap.begin();
		auto itEnd = mesh2PrimitivesMap.end();

		for (; it != itEnd; ++it) {
			auto p = (it->second).begin();	// each primitive
			auto pEnd = (it->second).end();
			for (; p != pEnd; ++p) {
				dim3 numBlocksForVertices((p->numVertices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);
				dim3 numBlocksForIndices((p->numIndices + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x);

				_vertexTransformAndAssembly<<<numBlocksForVertices, numThreadsPerBlock>>>(p->numVertices, *p, MVP, MV, MV_normal, width, height);
				checkCUDAError("Vertex Processing");
				hipDeviceSynchronize();
				_primitiveAssembly<<<numBlocksForIndices, numThreadsPerBlock>>>(
					p->numIndices, 
					curPrimitiveBeginId, 
					dev_primitives, 
					*p);
				checkCUDAError("Primitive Assembly");

				curPrimitiveBeginId += p->numPrimitives;
			}
		}

		checkCUDAError("Vertex Processing and Primitive Assembly");
	}

	//hipMemset(dev_fragmentBuffer, 0, width * height * sizeof(Fragment));
	//initDepth<<<blockCount2d, blockSize2d>>>(width, height, dev_depth);
	
	int numTiles = numTilesX * numTilesY;
	hipMemset(dev_primCounts, 0, numTiles * sizeof(int));

	// TODO: rasterize
	const int blockSize1d = 128;
	int numBlocks = (totalNumPrimitives + blockSize1d - 1) / blockSize1d;
	fillTileTriLists<<<numBlocks, blockSize1d>>>(totalNumPrimitives, numTilesX, numTilesY, triListSize, dev_primitives, dev_primCounts, dev_tileTriLists);

	dim3 numBlocks3(numTilesX, numTilesY, 1);
	dim3 blockSize3(TILE_SIZE, TILE_SIZE, 1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(tileBasedRasterize), hipFuncCachePreferL1);
	tileBasedRasterize<<<numBlocks3, blockSize3>>>(totalNumPrimitives, width, height, numTilesX, numTilesY, triListSize, dev_primitives, dev_primCounts, dev_tileTriLists, dev_depth, dev_fragmentBuffer);
	checkCUDAError("tileBasedRasterize");

	//_rasterize<<<numBlocks, blockSize1d>>>(totalNumPrimitives, width, height, dev_primitives, dev_depth, dev_fragmentBuffer);

    // Copy depthbuffer colors into framebuffer
	hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
	render<<<blockCount2d, blockSize2d>>>(width, height, dev_fragmentBuffer, dev_framebuffer);
	checkCUDAError("fragment shader");
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("copy render result to pbo");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

    // deconstruct primitives attribute/indices device buffer

	auto it(mesh2PrimitivesMap.begin());
	auto itEnd(mesh2PrimitivesMap.end());
	for (; it != itEnd; ++it) {
		for (auto p = it->second.begin(); p != it->second.end(); ++p) {
			hipFree(p->dev_indices);
			hipFree(p->dev_position);
			hipFree(p->dev_normal);
			hipFree(p->dev_texcoord0);
			hipFree(p->dev_diffuseTex);

			hipFree(p->dev_verticesOut);

			
			//TODO: release other attributes and materials
		}
	}

	////////////

    hipFree(dev_primitives);
    dev_primitives = NULL;

	hipFree(dev_fragmentBuffer);
	dev_fragmentBuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

	hipFree(dev_primCounts);
	dev_primCounts = nullptr;

	hipFree(dev_tileTriLists);
	dev_tileTriLists = nullptr;

    checkCUDAError("rasterize Free");
}
